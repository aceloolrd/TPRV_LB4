#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <chrono>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define BLOCK_SIZE 32 

void getInfoCUDADevice(hipDeviceProp_t& prop, int id) {
	printf("CUDA device %i Название   - %s\n", id, prop.name);
	printf("CUDA device %i Размер warp'а в потоках выполнения  - %i\n", id, prop.warpSize);
	printf("CUDA device %i Максимальное количество потоков выполнения в блоке  - %i\n", id, prop.maxThreadsPerBlock);
	printf("CUDA device %i Количество мультипроцессоров на устройстве  - %i\n", id, prop.multiProcessorCount);
	printf("CUDA device %i Максимальный размер каждого измерения блока потоков выполнения  - %i %i %i\n", id, prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("CUDA device %i Максимальный размер каждого измерения сетки блоков потоков выполнения  - %i %i %i\n", id, prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
}

__global__ void matrixMult(const __int16* Am, const __int16* Bm, __int16* result, int size) {
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	// вычисление индексов, используемых для доступа к элементам матриц и мас-сиву результата в ядре умножения матриц
	int ia = size * (gridDim.y * by + ty);
	int ib = gridDim.x * bx + tx;
	int ic = ia + ib;

	__int16 sum = 0;

	for (int k = 0; k < size; k++) {
		sum += Am[ia + k] * Bm[k * size + ib];
	}
	result[ic] = sum;
}


void compareMatrix(const __int16* f, const __int16* s, int size) {
	for (int i = 0; i < size; ++i) {
		for (int j = 0; j < size; ++j) {
			if (f[i * size + j] != s[i * size + j]) {
				printf("Matrixes not equal!\n");
				return;
			}
		}
	}
	printf("Matrices are equal!\n");
}

int main()
{
	setlocale(LC_ALL, "Russian");

	int count;
	hipDeviceProp_t prop;
	hipGetDeviceCount(&count);
	hipGetDeviceProperties(&prop, count - 1);
	getInfoCUDADevice(prop, count - 1);

	int size = 1024;

	size_t byte_size = size * size * sizeof(__int16);
	__int16* Am = (__int16*)malloc(byte_size);
	__int16* Bm = (__int16*)malloc(byte_size);
	__int16* GPU_C = (__int16*)malloc(byte_size);
	__int16* CPU_C = (__int16*)malloc(byte_size);

	for (int i = 0; i < size * size; ++i) {
		Am[i] = rand() % 6;
		Bm[i] = rand() % 6;
		CPU_C[i] = 0;
	}

	printf("\nScalar: \n");
	auto start = std::chrono::system_clock::now();
	for (int i = 0; i < size; ++i) {
		for (int j = 0; j < size; ++j) {
			for (int k = 0; k < size; ++k) {
				CPU_C[i * size + j] += Am[i * size + k] * Bm[k * size + j];
			}
		}
	}
	auto end = std::chrono::system_clock::now();
	std::chrono::duration<double> diff = end - start;

	printf("Time: %f seconds\n", diff);


	printf("GPU: \n");

	__int16* d_A = NULL;
	hipMalloc((void**)&d_A, byte_size);
	hipMemcpy(d_A, Am, byte_size, hipMemcpyHostToDevice);

	__int16* d_B = NULL;
	hipMalloc((void**)&d_B, byte_size);
	hipMemcpy(d_B, Bm, byte_size, hipMemcpyHostToDevice);

	__int16* d_C = NULL;
	hipMalloc((void**)&d_C, byte_size);


	hipEvent_t startEvent, stopEvent;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);

	hipEventRecord(startEvent, 0);
	const dim3 block(32, 32);
	const dim3 grid((size) / block.x, (size) / block.y);
	matrixMult << < grid, block >> > (d_A, d_B, d_C, size);

	// ждем завершения
	hipDeviceSynchronize();
	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, startEvent, stopEvent);

	hipMemcpy(GPU_C, d_C, byte_size, hipMemcpyDeviceToHost);

	printf("Time: %f seconds\n", milliseconds / 1000);
	compareMatrix(GPU_C, CPU_C, size);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(Am);
	free(Bm);
	free(GPU_C);
	free(CPU_C);
	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);

	return 0;
}
